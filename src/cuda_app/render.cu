#include "hip/hip_runtime.h"
#include "render.hpp"
#include "cuda_def.cuh"

constexpr int THREADS_PER_BLOCK = 1024;

constexpr int calc_thread_blocks(u32 n_threads)
{
    return (n_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}


class MandelbrotProps
{
public:
    u32 max_iter;
    r64 min_re;
    r64 min_im;
    r64 re_step;
    r64 im_step;
    DeviceMatrix iterations;
};


GPU_KERNAL
static void gpu_mandelbrot(MandelbrotProps props)
{
    auto const width = props.iterations.width;
    auto const height = props.iterations.height;
    auto n_elements = width * height;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    auto y = i / width;
    auto x = i - y * width;

    r64 const ci = props.min_im + y * props.im_step;
    u32 iter = 0;
    r64 const cr = props.min_re + x * props.re_step;

    r64 re = 0.0;
    r64 im = 0.0;
    r64 re2 = 0.0;
    r64 im2 = 0.0;

    while (iter < props.max_iter && re2 + im2 <= 4.0)
    {
        im = (re + re) * im + ci;
        re = re2 - im2 + cr;
        im2 = im * im;
        re2 = re * re;

        ++iter;
    }

    props.iterations.data[i] = props.iterations.data_mirror[i] = iter - 1;
}



GPU_KERNAL
static void gpu_set_color(DeviceImage image)
{
    auto const width = image.width;
    auto const height = image.height;
    auto n_pixels = width * height;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_pixels)
    {
        return;
    }

    // i = y * width + x

    pixel_t p = {};
    p.alpha = 255;
    p.red = 0;
    p.green = 0;
    p.blue = 0;

    auto y = i / width;
    auto x = i - y * width;

    if(y < height / 3)
    {
        p.red = 255;
    }
    else if(y < height * 2 / 3)
    {
        p.green = 255;
    }
    else
    {
        p.blue = 255;
    }

    if(x < width / 3)
    {
        p.red = 255;
    }
    else if(x < width * 2 / 3)
    {
        p.green = 255;
    }
    else
    {
        p.blue = 255;
    }

    image.data[i] = p;
}



GPU_FUNCTION
static u32 min_value(u32* sorted)
{
    return sorted[0];
}


GPU_FUNCTION
static u32 max_value(u32* sorted)
{
    return sorted[1];
}


GPU_KERNAL
static void gpu_sort_high_low(u32* values, u32 n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    u32* low = values;
    u32* high = values + n_elements / 2;

    if(high[i] < low[i])
    {
        auto h = low[i];
        low[i] = high[i];
        high[i] = h;
    }    
}


GPU_KERNAL
static void gpu_reduce_min_max(u32* values, u32 n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    auto half = n_elements / 2;

    if(i < half)
    {
        values[i] = min(values[i], values[i + 1]);
    }
    else
    {
        values[i] = max(values[i + half], values[i + half + 1]);
    }
}


HOST_FUNCTION
static void sort_min_max(DeviceMatrix& mat)
{
    u32 n_elements = mat.width * mat.height;

    assert(n_elements % 2 == 0);

    auto values = mat.data_mirror;

    bool proc = cuda_no_errors();
    assert(proc);

    gpu_sort_high_low<<<calc_thread_blocks(n_elements), THREADS_PER_BLOCK>>>(values, n_elements);

    proc &= cuda_launch_success();
    assert(proc);

    for(u32 n = n_elements / 2; n > 1; n /= 2)
    {
        gpu_reduce_min_max<<<calc_thread_blocks(n), THREADS_PER_BLOCK>>>(values, n);

        proc &= cuda_launch_success();
        assert(proc);
    }
}


void render(AppState& state)
{
    auto& d_screen = state.device.pixels;
    u32 n_pixels = d_screen.width * d_screen.height;
    int blocks = calc_thread_blocks(n_pixels);
    
    MandelbrotProps m_props{};
    m_props.max_iter = state.max_iter;
	m_props.min_re = MBT_MIN_X + state.mbt_pos.x;
	m_props.min_im = MBT_MIN_Y + state.mbt_pos.y;
	m_props.re_step = state.mbt_screen_width / d_screen.width;
	m_props.im_step = state.mbt_screen_height / d_screen.height;
    m_props.iterations = state.device.iterations;    

    bool proc = cuda_no_errors();
    assert(proc);

    gpu_mandelbrot<<<blocks, THREADS_PER_BLOCK>>>(m_props);

    proc &= cuda_launch_success();
    assert(proc);

    sort_min_max(state.device.iterations);

    gpu_set_color<<<blocks, THREADS_PER_BLOCK>>>(d_screen);

    proc &= cuda_launch_success();
    assert(proc);

    auto& h_screen = state.screen_buffer;
    proc &= copy_to_host(d_screen, h_screen);
    assert(proc);
}