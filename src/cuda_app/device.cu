#include "device.hpp"
#include "cuda_def.cuh"

#include <cassert>

#ifdef CUDA_PRINT_ERROR

#include <cstdio>
#include <cstring>

#endif


static void check_error(hipError_t err, cstr label = "")
{
    if(err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR
    #ifndef	NDEBUG

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));

    if(std::strlen(label))
    {
        printf("\n%s", label);
    }
    
    printf("\n\n******************\n\n");

    #endif
    #endif
}


namespace cuda
{
    bool device_malloc(DevicePointer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if(!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err, "device_malloc");

        bool result = err == hipSuccess;

        assert(result);

        return result;
    }


    bool unified_malloc(DevicePointer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if(!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMallocManaged((void**)&(buffer.data), n_bytes);
        check_error(err, "unified_malloc");

        bool result = err == hipSuccess;

        assert(result);

        return result;
    }


    bool free(void* data)
    {
        if(data)
        {
            return true;
        }

        hipError_t err = hipFree(data);
        check_error(err, "free");

        return err == hipSuccess;
    }


    bool memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
    {
        hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
        check_error(err, "memcpy_to_device");

        bool result = err == hipSuccess;

        assert(result);

        return result;
    }


    bool memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
    {
        hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
        check_error(err, "memcpy_to_host");

        bool result = err == hipSuccess;

        assert(result);

        return result;
    }


    bool no_errors(cstr label)
    {
        hipError_t err = hipGetLastError();
        check_error(err, label);

        return err == hipSuccess;
    }


    bool launch_success(cstr label)
    {
        hipError_t err = hipDeviceSynchronize();
        check_error(err, label);

        return err == hipSuccess;
    }
}