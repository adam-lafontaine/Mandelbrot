#include "device.hpp"
#include "cuda_def.cuh"

#ifdef CUDA_PRINT_ERROR

#include <cstdio>

#endif


static void check_error(hipError_t err)
{
    if(err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));
    printf("\n\n******************\n\n");

    #endif
}

/*
static bool cuda_device_malloc(void** ptr, u32 n_bytes)
{
    hipError_t err = hipMalloc(ptr, n_bytes);
    check_error(err);
    
    return err == hipSuccess;
}


static bool cuda_unified_malloc(void** ptr, u32 n_bytes)
{
    hipError_t err = hipMallocManaged(ptr, n_bytes);
    check_error(err);
    
    return err == hipSuccess;
}


static bool cuda_device_free(void* ptr)
{
    hipError_t err = hipFree(ptr);
    check_error(err);

    return err == hipSuccess;
}
*/


bool cuda_memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_no_errors()
{
    hipError_t err = hipGetLastError();
    check_error(err);

    return err == hipSuccess;
}


bool cuda_launch_success()
{
    hipError_t err = hipDeviceSynchronize();
    check_error(err);

    return err == hipSuccess;
}

/*
bool device_malloc(DeviceBuffer& buffer, size_t n_bytes)
{
    bool result = cuda_device_malloc((void**)&(buffer.data), n_bytes);
    if(result)
    {
        buffer.total_bytes = n_bytes;
    }

    return result;
}
*/

/*
bool unified_malloc(DeviceBuffer& buffer, size_t n_bytes)
{
    bool result = cuda_unified_malloc((void**)&(buffer.data), n_bytes);
    if(result)
    {
        buffer.total_bytes = n_bytes;
    }

    return result;
}


bool device_free(DeviceBuffer& buffer)
{
    buffer.total_bytes = 0;
    buffer.offset = 0;
    return cuda_device_free(buffer.data);
}
*/

/*
bool make_device_image(DeviceImage& image, u32 width, u32 height, DeviceBuffer& buffer)
{
    assert(buffer.data);
    auto bytes = width * height * sizeof(pixel_t);

    bool result = buffer.total_bytes - buffer.offset >= bytes;
    if(result)
    {
        image.width = width;
        image.height = height;
        image.data = (pixel_t*)(buffer.data + buffer.offset);
        buffer.offset += bytes;
    }

    return result;
}
*/

bool copy_to_device(image_t const& src, DeviceImage const& dst)
{
    assert(src.data);
    assert(src.width);
    assert(src.height);
    assert(dst.data);
    assert(dst.width == src.width);
    assert(dst.height == src.height);

    auto bytes = src.width * src.height * sizeof(pixel_t);

    return cuda_memcpy_to_device(src.data, dst.data, bytes);
}


bool copy_to_host(DeviceImage const& src, image_t const& dst)
{
    assert(src.data);
    assert(src.width);
    assert(src.height);
    assert(dst.data);
    assert(dst.width == src.width);
    assert(dst.height == src.height);

    auto bytes = src.width * src.height * sizeof(pixel_t);

    return cuda_memcpy_to_host(src.data, dst.data, bytes);
}

/*
bool make_device_matrix(DeviceMatrix& matrix, u32 width, u32 height, DeviceBuffer& buffer)
{
    assert(buffer.data);
    auto bytes_per = width * height * sizeof(u32);

    bool result = buffer.total_bytes - buffer.offset >= 2 * bytes_per;
    if(result)
    {
        matrix.width = width;
        matrix.height = height;
        matrix.data_src = (u32*)(buffer.data + buffer.offset);
        buffer.offset += bytes_per;
        matrix.data_dst = (u32*)(buffer.data + buffer.offset);
        buffer.offset += bytes_per;
    }

    return result;
}
*/

/*
bool make_device_palette(DeviceColorPalette& palette, u32 n_colors, DeviceBuffer& buffer)
{
    assert(buffer.data);

    auto bytes_per_channel = sizeof(u8) * n_colors;
    auto bytes = RGB_CHANNELS * bytes_per_channel;

    bool result = buffer.total_bytes - buffer.offset >= bytes;

    if(!result)
    {
        return false;
    }

    palette.n_colors = n_colors;

    for(u32 c = 0; c < RGB_CHANNELS; ++c)
    {
        palette.channels[c] = buffer.data + buffer.offset;
        buffer.offset += bytes_per_channel;
    }

    return result;
}
*/


namespace device
{
    bool malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if(!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err);

        bool result = err == hipSuccess;

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool unified_malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if(!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMallocManaged((void**)&(buffer.data), n_bytes);
        check_error(err);

        bool result = err == hipSuccess;

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool free(MemoryBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.size = 0;

        if(buffer.data)
        {
            hipError_t err = hipFree(buffer.data);
            check_error(err);

            buffer.data = nullptr;

            return err == hipSuccess;
        }

        return true;
    }


    u8* push_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size < buffer.capacity);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size < buffer.capacity;

        auto bytes_available = (buffer.capacity - buffer.size) >= n_bytes;
        assert(bytes_available);

        if(!is_valid || !bytes_available)
        {
            return nullptr;
        }

        auto data = buffer.data + buffer.size;

        buffer.size += n_bytes;

        return data;
    }


    bool pop_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size <= buffer.capacity);
        assert(n_bytes <= buffer.capacity);
        assert(n_bytes <= buffer.size);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size <= buffer.capacity &&
            n_bytes <= buffer.capacity &&
            n_bytes <= buffer.size;

        if(is_valid)
        {
            buffer.size -= n_bytes;
            return true;
        }

        return false;
    }


    bool push_device_image(MemoryBuffer& buffer, DeviceImage& image, u32 width, u32 height)
    {
        auto data = push_bytes(buffer, width * height * sizeof(Pixel));

        if(data)
        {
            image.width = width;
            image.height = height;
            image.data = (Pixel*)data;

            return true;
        }

        return false;
    }


    bool push_device_matrix(MemoryBuffer& buffer, DeviceMatrix& matrix, u32 width, u32 height)
    {
        auto bytes_per = width * height * sizeof(u32);
        auto src_data = push_bytes(buffer, bytes_per);

        if(!src_data)
        {
            return false;
        }

        auto dst_data = push_bytes(buffer, bytes_per);
        if(!dst_data)
        {
            pop_bytes(buffer, bytes_per);
            return false;
        }

        matrix.width = width;
        matrix.height = height;
        matrix.data_src = (u32*)src_data;
        matrix.data_dst = (u32*)dst_data;

        return true;
    }


    bool push_device_palette(MemoryBuffer& buffer, DeviceColorPalette& palette, u32 n_colors)
    {
        auto bytes_per_channel = sizeof(u8) * n_colors;
        size_t bytes_allocated = 0;

        for(u32 c = 0; c < RGB_CHANNELS; ++c)
        {
            auto data = push_bytes(buffer, bytes_per_channel);
            if(!data)
            {
                break;                
            }

            bytes_allocated += bytes_per_channel;
            palette.channels[c] = (u8*)data;
        }

        if(bytes_allocated == RGB_CHANNELS * bytes_per_channel)
        {
            palette.n_colors = n_colors;
            return true;
        }
        else if (bytes_allocated > 0)
        {
            pop_bytes(buffer, bytes_allocated);            
        }

        return false;
    }
}
